#include "hip/hip_runtime.h"
#include "cuda_defines.h"
#include "cuda_insertsource.h"

__global__ void kernel_InsertSource(const float val, const int iSource,
	                            float * restrict qp, float * restrict qc)
{
  const int ix=blockIdx.x * blockDim.x + threadIdx.x;
  if (ix==0)
  {
    qp[iSource]+=val;
    qc[iSource]+=val;
  }
}


void CUDA_InsertSource(const float val, const int iSource, float * restrict pc, float * restrict qc,  float * restrict pp, float * restrict qp)
{

  extern float* dev_pp;
  extern float* dev_pc;
  extern float* dev_qp;
  extern float* dev_qc;

  if ((pp) && (qp))
  {
     dim3 threadsPerBlock(BSIZE_X, 1);
     dim3 numBlocks(1,1);
     kernel_InsertSource<<<numBlocks, threadsPerBlock>>> (val, iSource, pc, qc);
     CUDA_CALL(hipGetLastError());
     CUDA_CALL(hipDeviceSynchronize());
  }
}
