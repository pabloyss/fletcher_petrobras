#include "hip/hip_runtime.h"
#include "cuda_defines.h"
#include "../driver.h"
#include "cuda_insertsource.h"

__global__ void kernel_InsertSource(const float val, const int iSource,
	                            float * restrict qp, float * restrict qc, int fix_position, int offset)
{
  const int ix=blockIdx.x * blockDim.x + threadIdx.x;
  if (ix==0)
  {
    qp[iSource - fix_position + offset]+=val;
    qc[iSource - fix_position + offset]+=val;
  }
}


void CUDA_InsertSource(const float val, const int iSource, float * restrict pc, float * restrict qc,  float * restrict pp, float * restrict qp)
{
  extern Gpu gpu_map[GPU_NUMBER];
  extern float* dev_pp[GPU_NUMBER];
  extern float* dev_pc[GPU_NUMBER];
  extern float* dev_qp[GPU_NUMBER];
  extern float* dev_qc[GPU_NUMBER];

  int num_gpus;
  int teste;
  int offset = 0;
  int fix_position = 0;
  CUDA_CALL(hipGetDeviceCount(&num_gpus));
  for (int gpu = 0; gpu < 2; gpu++)
    {
        hipDeviceProp_t prop;
        hipSetDevice(gpu);
        CUDA_CALL(hipGetDeviceProperties(&prop, gpu));
        if ((dev_pp[gpu]) && (dev_qp[gpu]))
        {
          if(gpu != 0){
            fix_position = gpu_map[0].cpu_end_pointer;
            offset = gpu_map[1].gpu_start_pointer;
            teste = iSource - fix_position + offset;
          }else{
            teste = iSource - fix_position + offset;
          }
          dim3 threadsPerBlock(BSIZE_X, 1);
          dim3 numBlocks(1,1);
          kernel_InsertSource<<<numBlocks, threadsPerBlock>>> (val, iSource, dev_pc[gpu], dev_qc[gpu], fix_position, offset);
        }
    }
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
}
