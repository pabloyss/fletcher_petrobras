#include "cuda_defines.h"
#include "cuda_stuff.h"
#include "../map.h"

void CUDA_Initialize(const int sx, const int sy, const int sz, const int bord,
                     float dx, float dy, float dz, float dt,
                     float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                     float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                     float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                     float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                     float *restrict phi, float *restrict theta,
                     float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{

   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));

   // Check sx, sy values
   if (sx % BSIZE_X != 0)
   {
      printf("sx(%d) must be multiple of BSIZE_X(%d)\n", sx, (int)BSIZE_X);
      exit(1);
   }
   if (sy % BSIZE_Y != 0)
   {
      printf("sy(%d) must be multiple of BSIZE_Y(%d)\n", sy, (int)BSIZE_Y);
      exit(1);
   }

   int sxsy = sx * sy; // one plan
   const size_t sxsysz = sxsy * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sxsy * sizeof(float); // 2 extra plans for wave fields

   const int strideX = ind(1, 0, 0) - ind(0, 0, 0);
   const int strideY = ind(0, 1, 0) - ind(0, 0, 0);
   const int strideZ = ind(0, 0, 1) - ind(0, 0, 0);

   for (int device = 0; device < deviceCount; device++)
   {
      hipDeviceProp_t deviceProp;
      CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
      printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
      CUDA_CALL(hipSetDevice(device));

      // arthur -- Se for realizar a cópia assíncrona com prefetch, é aqui o lugar.

      int gpuLower, gpuUpper, numElements;
      if (device == 0)
      {
         gpuLower = 0;
         gpuUpper = (sz / 2) + 5;  // +4 devido ao dercross que usar 4*stride (+ e -) 
         numElements = (sx*sy*gpuUpper) - (sx*sy*gpuLower);
      }
      else
      {
         gpuLower = (sz / 2 ) - 5;  // -4 devido ao dercross que usar 4*stride (+ e -) 
         gpuUpper = sz;
         numElements = (sx*sy*gpuUpper) - (sx*sy*gpuLower);
      }

       // Prefetch necessary arrays for the GPU

      hipMemPrefetchAsync(&ch1dxx[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&ch1dyy[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&ch1dzz[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&ch1dxy[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&ch1dyz[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&ch1dxz[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&v2px[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&v2pz[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&v2sz[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&v2pn[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&pp[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&pc[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&qp[gpuLower], numElements * sizeof(float), device);
      hipMemPrefetchAsync(&qc[gpuLower], numElements * sizeof(float), device);

//}
      //CUDA_CALL(hipDeviceSynchronize());
      //CUDA_CALL(hipGetLastError());

      printf("GPU memory usage = %ld MiB\n", 15 * msize_vol / 1024 / 1024);

      size_t freeMem, totalMem;
      CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
      printf("GPU memory usage: %lu MiB (used) / %lu MiB (total)\n", (totalMem - freeMem) / (1024 * 1024), totalMem / (1024 * 1024));
   }
   pp += sxsy;
   pc += sxsy;
   qp += sxsy;
   qc += sxsy;
}

// ARTHUR - Ajustar função para receber os parametros do CUDA_Finalize.
void CUDA_Finalize(const int sx, const int sy, const int sz, const int bord,
                   float dx, float dy, float dz, float dt,
                   float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                   float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                   float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                   float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                   float *restrict phi, float *restrict theta,
                   float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{
   int sxsy = sx * sy; // one plan
   pp -= sxsy;
   pc -= sxsy;
   qp -= sxsy;
   qc -= sxsy;

   CUDA_CALL(hipFree(vpz));
   CUDA_CALL(hipFree(vsv));
   CUDA_CALL(hipFree(epsilon));
   CUDA_CALL(hipFree(delta));
   CUDA_CALL(hipFree(phi));
   CUDA_CALL(hipFree(theta));
   CUDA_CALL(hipFree(ch1dxx));
   CUDA_CALL(hipFree(ch1dyy));
   CUDA_CALL(hipFree(ch1dzz));
   CUDA_CALL(hipFree(ch1dxy));
   CUDA_CALL(hipFree(ch1dyz));
   CUDA_CALL(hipFree(ch1dxz));
   CUDA_CALL(hipFree(v2px));
   CUDA_CALL(hipFree(v2pz));
   CUDA_CALL(hipFree(v2sz));
   CUDA_CALL(hipFree(v2pn));
   // CUDA_CALL(hipFree(pp));
   // CUDA_CALL(hipFree(pc));
   // CUDA_CALL(hipFree(qp));
   // CUDA_CALL(hipFree(qc));

   printf("CUDA_Finalize: SUCCESS\n");
}

void CUDA_Update_pointers(const int sx, const int sy, const int sz, float *pc)
{
   // arthur: se usar memoria unificada, não precisa desta cópia.
   // extern float* dev_pc;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   // if (pc) CUDA_CALL(hipMemcpy(pc, dev_pc, msize_vol, hipMemcpyDeviceToHost));
   CUDA_CALL(hipMemPrefetchAsync(pc, msize_vol, hipCpuDeviceId));
}

void CUDA_prefetch_pc(const int sx, const int sy, const int sz, float *pc){

   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));

   for (int device = 0; device < deviceCount; device++)
   {
      hipDeviceProp_t deviceProp;
      CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
      printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
      CUDA_CALL(hipSetDevice(device));

      // arthur -- Se for realizar a cópia assíncrona com prefetch, é aqui o lugar.

      int gpuLower, gpuUpper, numElements;
      if (device == 0)
      {
         gpuLower = 0;
         gpuUpper = (sz / 2) + 5;  // +4 devido ao dercross que usar 4*stride (+ e -) 
         numElements = (sx*sy*gpuUpper) - (sx*sy*gpuLower);
      }
      else
      {
         gpuLower = (sz / 2 ) - 5;  // -4 devido ao dercross que usar 4*stride (+ e -) 
         gpuUpper = sz;
         numElements = (sx*sy*gpuUpper) - (sx*sy*gpuLower);
      }

      hipMemPrefetchAsync(&pc[gpuLower], numElements * sizeof(float), device);
   }
}

void CUDA_Allocate_Model_Variables(float **restrict ch1dxx, float **restrict ch1dyy, float **restrict ch1dzz, float **restrict ch1dxy,
                                   float **restrict ch1dyz, float **restrict ch1dxz, float **restrict v2px, float **restrict v2pz, float **restrict v2sz,
                                   float **restrict v2pn, int sx, int sy, int sz)
{
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   CUDA_CALL(hipMallocManaged(ch1dxx, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dyy, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dzz, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dxy, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dyz, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dxz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2px, msize_vol));
   CUDA_CALL(hipMallocManaged(v2pz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2sz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2pn, msize_vol));
}

void CUDA_Allocate_main(float **restrict vpz, float **restrict vsv, float **restrict epsilon, float **restrict delta,
                        float **restrict phi, float **restrict theta, float **restrict pp, float **restrict pc, float **restrict qp,
                        float **restrict qc, int sx, int sy, int sz)
{
   int sxsy = sx * sy;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sx * sy * sizeof(float); // 2 extra plans for wave fields
   CUDA_CALL(hipMallocManaged(vpz, msize_vol));
   CUDA_CALL(hipMallocManaged(vsv, msize_vol));
   CUDA_CALL(hipMallocManaged(epsilon, msize_vol));
   CUDA_CALL(hipMallocManaged(delta, msize_vol));
   CUDA_CALL(hipMallocManaged(phi, msize_vol));
   CUDA_CALL(hipMallocManaged(theta, msize_vol));

   CUDA_CALL(hipMallocManaged(pp, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(pc, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(qp, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(qc, msize_vol_extra));
   // ARTHUR - Ver se esta operação fica na CPU ou mover para a GPU.
   memset(*pp, 0, msize_vol_extra);
   memset(*pc, 0, msize_vol_extra);
   memset(*qp, 0, msize_vol_extra);
   memset(*qc, 0, msize_vol_extra);
   // pp+=sxsy;
   // pc+=sxsy;
   // qp+=sxsy;
   // qc+=sxsy;
}