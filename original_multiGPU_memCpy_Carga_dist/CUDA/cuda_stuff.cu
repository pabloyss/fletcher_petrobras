#include "cuda_defines.h"
#include "cuda_stuff.h"
#include "../driver.h"
#include "../map.h"

void CUDA_Initialize(const int sx, const int sy, const int sz, const int bord,
                     float dx, float dy, float dz, float dt,
                     float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                     float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                     float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                     float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                     float *restrict phi, float *restrict theta,
                     float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{


   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));

   
   extern float* dev_ch1dxx[GPU_NUMBER];
   extern float* dev_ch1dyy[GPU_NUMBER];
   extern float* dev_ch1dzz[GPU_NUMBER];
   extern float* dev_ch1dxy[GPU_NUMBER];
   extern float* dev_ch1dyz[GPU_NUMBER];
   extern float* dev_ch1dxz[GPU_NUMBER];
   extern float* dev_v2px[GPU_NUMBER];
   extern float* dev_v2pz[GPU_NUMBER];
   extern float* dev_v2sz[GPU_NUMBER];
   extern float* dev_v2pn[GPU_NUMBER];
   extern float* dev_pp[GPU_NUMBER];
   extern float* dev_pc[GPU_NUMBER];
   extern float* dev_qp[GPU_NUMBER];
   extern float* dev_qc[GPU_NUMBER];


   // Check sx, sy values
   if (sx % BSIZE_X != 0)
   {
      printf("sx(%d) must be multiple of BSIZE_X(%d)\n", sx, (int)BSIZE_X);
      exit(1);
   }
   if (sy % BSIZE_Y != 0)
   {
      printf("sy(%d) must be multiple of BSIZE_Y(%d)\n", sy, (int)BSIZE_Y);
      exit(1);
   }

   int sxsy = sx * sy; // one plan
   const size_t sxsysz = sxsy * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sxsy * sizeof(float); // 2 extra plans for wave fields

   const int strideX = ind(1, 0, 0) - ind(0, 0, 0);
   const int strideY = ind(0, 1, 0) - ind(0, 0, 0);
   const int strideZ = ind(0, 0, 1) - ind(0, 0, 0);

   // Cópia dos dados para cada GPU
   for (int device = 0; device < 2; device++)
   {
      hipDeviceProp_t deviceProp;
      CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
      printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
      CUDA_CALL(hipSetDevice(device));

      const size_t sxsysz = ((size_t)sx * sy) * sz;
      const size_t msize_vol = sxsysz * sizeof(float);
      const size_t msize_vol_extra = msize_vol + 2 * sx*sy * sizeof(float); // 2 extra plans for wave fields
      const size_t msize_vol_half = msize_vol_extra / 2;
      const int size_space = (ind(0, 0 , sz/2) - ind(0, 0, (sz/2 - 4))) * sizeof(float);
      const int size_bord = ind(0, 0, (sz / 2));
      const int size_lower = ind(0,0,0);
      const int size_gpu0 = ind(0,0,(sz/2 - 4));
      const int size_gpu1 = ind(sx,sy,(sz/2 + 4));
      const int size_offset = (size_gpu0 - size_lower);
      const int size_swap_gpu0 = size_bord - ind(0, 0, (sz/2 - 4));
      const int size_swap_gpu1 = ind(sx,sy, (sz/2 +4)) - size_bord;
      const int size_total_gpu1 = (ind(sz,sy,(sz)) - ind(0, 0, (sz/2 - 4))) * sizeof(float);
      const int size_total_gpu0 = (size_gpu1 - ind(0,0,0)) * sizeof(float);
      if (device == 0){
         CUDA_CALL(hipMalloc(&dev_ch1dxx[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_ch1dyy[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_ch1dzz[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_ch1dxy[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_ch1dyz[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_ch1dxz[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_v2px[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_v2pz[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_v2sz[device], size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_v2pn[device], size_total_gpu0));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dxx[device], ch1dxx, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dyy[device], ch1dyy, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dzz[device], ch1dzz, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dxy[device], ch1dxy, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dyz[device], ch1dyz, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_ch1dxz[device], ch1dxz, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_v2px[device], v2px, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_v2pz[device], v2pz, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_v2sz[device], v2sz, size_total_gpu0, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpyAsync(dev_v2pn[device], v2pn, size_total_gpu0, hipMemcpyHostToDevice));

         // Wave field arrays with an extra plan
         CUDA_CALL(hipMalloc(&dev_pp[device], size_total_gpu0));
         CUDA_CALL(hipMemset(dev_pp[device], 0, size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_pc[device], size_total_gpu0));
         CUDA_CALL(hipMemset(dev_pc[device], 0, size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_qp[device], size_total_gpu0));
         CUDA_CALL(hipMemset(dev_qp[device], 0, size_total_gpu0));
         CUDA_CALL(hipMalloc(&dev_qc[device], size_total_gpu0));
         CUDA_CALL(hipMemset(dev_qc[device], 0, size_total_gpu0));

      }else{

         CUDA_CALL(hipMalloc(&dev_ch1dxx[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_ch1dyy[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_ch1dzz[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_ch1dxy[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_ch1dyz[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_ch1dxz[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_v2px[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_v2pz[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_v2sz[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_v2pn[device], size_total_gpu1));

         CUDA_CALL(hipMemcpy(dev_ch1dxx[device], ch1dxx + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_ch1dyy[device], ch1dyy + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_ch1dzz[device], ch1dzz + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_ch1dxy[device], ch1dxy + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_ch1dyz[device], ch1dyz + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_ch1dxz[device], ch1dxz + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_v2px[device], v2px + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_v2pz[device], v2pz + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_v2sz[device], v2sz + size_offset, size_total_gpu1, hipMemcpyHostToDevice));
         CUDA_CALL(hipMemcpy(dev_v2pn[device], v2pn + size_offset, size_total_gpu1, hipMemcpyHostToDevice));

         CUDA_CALL(hipMalloc(&dev_pp[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_pc[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_qp[device], size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_qc[device], size_total_gpu1));

         // Wave field arrays with an extra plan
         CUDA_CALL(hipMalloc(&dev_pp[device], size_total_gpu1));
         CUDA_CALL(hipMemset(dev_pp[device], 0, size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_pc[device], size_total_gpu1));
         CUDA_CALL(hipMemset(dev_pc[device], 0, size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_qp[device], size_total_gpu1));
         CUDA_CALL(hipMemset(dev_qp[device], 0, size_total_gpu1));
         CUDA_CALL(hipMalloc(&dev_qc[device], size_total_gpu1));
         CUDA_CALL(hipMemset(dev_qc[device], 0, size_total_gpu1));
      }

     // dev_pp[device]+=sxsy;
     // dev_pc[device]+=sxsy;
     // dev_qp[device]+=sxsy;
     // dev_qc[device]+=sxsy;


      printf("GPU memory usage = %ld MiB\n", 15 * msize_vol / 1024 / 1024);
      size_t freeMem, totalMem;
      CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
      printf("GPU memory usage: %lu MiB (used) / %lu MiB (total)\n", (totalMem - freeMem) / (1024 * 1024), totalMem / (1024 * 1024));
   }
   CUDA_CALL(hipDeviceSynchronize());
   CUDA_CALL(hipGetLastError());
}
  

// ARTHUR - Ajustar função para receber os parametros do CUDA_Finalize.
void CUDA_Finalize(const int sx, const int sy, const int sz, const int bord,
                   float dx, float dy, float dz, float dt,
                   float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                   float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                   float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                   float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                   float *restrict phi, float *restrict theta,
                   float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{


   extern float* dev_ch1dxx[GPU_NUMBER];
   extern float* dev_ch1dyy[GPU_NUMBER];
   extern float* dev_ch1dzz[GPU_NUMBER];
   extern float* dev_ch1dxy[GPU_NUMBER];
   extern float* dev_ch1dyz[GPU_NUMBER];
   extern float* dev_ch1dxz[GPU_NUMBER];
   extern float* dev_v2px[GPU_NUMBER];
   extern float* dev_v2pz[GPU_NUMBER];
   extern float* dev_v2sz[GPU_NUMBER];
   extern float* dev_v2pn[GPU_NUMBER];
   extern float* dev_pp[GPU_NUMBER];
   extern float* dev_pc[GPU_NUMBER];
   extern float* dev_qp[GPU_NUMBER];
   extern float* dev_qc[GPU_NUMBER];
   extern float* bordSwap[GPU_NUMBER];

   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));
   int sxsy = sx * sy; // one plan
   for (int device = 0; device < 2; device++)
   {
      hipDeviceProp_t deviceProp;
      CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
      printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
      CUDA_CALL(hipSetDevice(device));


   dev_pp[device] -= sxsy;
   dev_pc[device] -= sxsy;
   dev_qp[device] -= sxsy;
   dev_qc[device] -= sxsy;

   CUDA_CALL(hipFree(dev_ch1dxx[device]));
   CUDA_CALL(hipFree(dev_ch1dyy[device]));
   CUDA_CALL(hipFree(dev_ch1dzz[device]));
   CUDA_CALL(hipFree(dev_ch1dxy[device]));
   CUDA_CALL(hipFree(dev_ch1dyz[device]));
   CUDA_CALL(hipFree(dev_ch1dxz[device]));
   CUDA_CALL(hipFree(dev_v2px[device]));
   CUDA_CALL(hipFree(dev_v2pz[device]));
   CUDA_CALL(hipFree(dev_v2sz[device]));
   CUDA_CALL(hipFree(dev_v2pn[device]));
   CUDA_CALL(hipFree(dev_pp[device]));
   CUDA_CALL(hipFree(dev_pc[device]));
   CUDA_CALL(hipFree(dev_qp[device]));
   CUDA_CALL(hipFree(dev_qc[device]));

   }

   printf("CUDA_Finalize: SUCCESS\n");
}
/*
void CUDA_Update_pointers(const int sx, const int sy, const int sz, float *pc)
{
   // arthur: se usar memoria unificada, não precisa desta cópia.
   extern float* dev_pc;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   if (pc){
      CUDA_CALL(hipMemcpyAsync(pc, dev_pc, msize_vol, hipMemcpyDeviceToHost));
   } 
}
*/
void CUDA_Update_pointers(const int sx, const int sy, const int sz, float* pc)
{
    extern float* dev_pc[GPU_NUMBER];
    int deviceCount;
    CUDA_CALL(hipGetDeviceCount(&deviceCount));
    const size_t sxsysz = ((size_t)sx * sy) * sz;
    const size_t msize_vol = sxsysz * sizeof(float);
    const size_t msize_vol_half = msize_vol / 2;

    for (int device = 0; device < 2; device++)
    {
        CUDA_CALL(hipSetDevice(device));

        if (device == 0)
        {
            // Copiar a primeira metade do array dev_pc[0] --> primeira metade do array pc
            CUDA_CALL(hipMemcpy(pc, dev_pc[0], msize_vol_half, hipMemcpyDeviceToHost));

        }
        else
        {
            // Copiar a segunda metade do array dev_pc[device] --> segunda metade do array pc
            CUDA_CALL(hipMemcpy(pc + (msize_vol_half / sizeof(float)), dev_pc[device] + (msize_vol_half / sizeof(float)), msize_vol_half, hipMemcpyDeviceToHost));

        }
        CUDA_CALL(hipDeviceSynchronize()); 
    }
}


void CUDA_prefetch_pc(const int sx, const int sy, const int sz, float *pc)
{

   extern float* dev_pc[GPU_NUMBER];
   int sxsy = sx * sy; // one plan
   const size_t sxsysz = sxsy * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sxsy * sizeof(float); // 2 extra plans for wave fields
   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));

   // Cálculo do número de elementos para cada GPU
   int numElementsPerGPU = (sx * sy * sz) / deviceCount;

   // Cálculo do número de elementos para as variáveis dev_pp, dev_pc, dev_qp e dev_qc
   int numExtraElementsPerGPU = msize_vol_extra / deviceCount;

   // Cópia dos dados para cada GPU
   for (int device = 0; device < 2; device++)
   {
   
      hipDeviceProp_t deviceProp;
      CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
      printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
      CUDA_CALL(hipSetDevice(device));

      // Calcula o intervalo de elementos para a GPU atual
      int gpuLower = device * numElementsPerGPU;
      int gpuUpper = (device == deviceCount - 1) ? (sx * sy * sz) : ((device + 1) * numElementsPerGPU);

      // Calcula o número de elementos para a GPU atual
      int numElements = gpuUpper - gpuLower;

      // Calcula o intervalo de elementos para as variáveis dev_pp, dev_pc, dev_qp e dev_qc
      int extraGpuLower = device * numExtraElementsPerGPU;
      int extraGpuUpper = (device == deviceCount - 1) ? msize_vol_extra : ((device + 1) * numExtraElementsPerGPU);

      // Calcula o número de elementos para as variáveis dev_pp, dev_pc, dev_qp e dev_qc
      int numExtraElements = extraGpuUpper - extraGpuLower;

      // Copia os dados da CPU para a GPU atual
      CUDA_CALL(hipMemcpyAsync(pc, dev_pc, msize_vol, hipMemcpyHostToDevice));
   }
}

void CUDA_Allocate_Model_Variables(float **restrict ch1dxx, float **restrict ch1dyy, float **restrict ch1dzz, float **restrict ch1dxy,
                                   float **restrict ch1dyz, float **restrict ch1dxz, float **restrict v2px, float **restrict v2pz, float **restrict v2sz,
                                   float **restrict v2pn, int sx, int sy, int sz)
{
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   CUDA_CALL(hipHostMalloc(ch1dxx, msize_vol));
   CUDA_CALL(hipHostMalloc(ch1dyy, msize_vol));
   CUDA_CALL(hipHostMalloc(ch1dzz, msize_vol));
   CUDA_CALL(hipHostMalloc(ch1dxy, msize_vol));
   CUDA_CALL(hipHostMalloc(ch1dyz, msize_vol));
   CUDA_CALL(hipHostMalloc(ch1dxz, msize_vol));
   CUDA_CALL(hipHostMalloc(v2px, msize_vol));
   CUDA_CALL(hipHostMalloc(v2pz, msize_vol));
   CUDA_CALL(hipHostMalloc(v2sz, msize_vol));
   CUDA_CALL(hipHostMalloc(v2pn, msize_vol));
}

void CUDA_Allocate_main(float **restrict vpz, float **restrict vsv, float **restrict epsilon, float **restrict delta,
                        float **restrict phi, float **restrict theta, float **restrict pp, float **restrict pc, float **restrict qp,
                        float **restrict qc, int sx, int sy, int sz)
{
   int sxsy = sx * sy;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sx * sy * sizeof(float); // 2 extra plans for wave fields
   CUDA_CALL(hipHostMalloc(vpz, msize_vol));
   CUDA_CALL(hipHostMalloc(vsv, msize_vol));
   CUDA_CALL(hipHostMalloc(epsilon, msize_vol));
   CUDA_CALL(hipHostMalloc(delta, msize_vol));
   CUDA_CALL(hipHostMalloc(phi, msize_vol));
   CUDA_CALL(hipHostMalloc(theta, msize_vol));

   CUDA_CALL(hipHostMalloc(pp, msize_vol_extra));
   CUDA_CALL(hipHostMalloc(pc, msize_vol_extra));
   CUDA_CALL(hipHostMalloc(qp, msize_vol_extra));
   CUDA_CALL(hipHostMalloc(qc, msize_vol_extra));
   // ARTHUR - Ver se esta operação fica na CPU ou mover para a GPU.
   memset(*pp, 0, msize_vol_extra);
   memset(*pc, 0, msize_vol_extra);
   memset(*qp, 0, msize_vol_extra);
   memset(*qc, 0, msize_vol_extra);
   // pp+=sxsy;
   // pc+=sxsy;
   // qp+=sxsy;
   // qc+=sxsy;
}